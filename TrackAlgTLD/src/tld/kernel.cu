#include "hip/hip_runtime.h"
#include "kernel.h"

using namespace std;
using namespace cv;
using namespace cv::gpu;

struct refInfo {
	int numWindows;
	bool isActive;
	int fgListSize;
	int truePositiveSize;
	int falsePositiveSize;
};

struct proData {
	int window[TLD_WINDOW_SIZE];

	// varianceFilter
	int varianceWindowOffsets[TLD_WINDOW_OFFSET_SIZE];

	// ensembleClassifier
	int ensembleWindowOffsets[TLD_WINDOW_OFFSET_SIZE];
	int featureVector[TLD_NUM_TREES];
	unsigned char subImgData[TLD_PATCH_SIZE * TLD_PATCH_SIZE];
	size_t step;
};

__global__ void detection(refInfo* info, proData *data, float *falsePositives,
		float *truePositives, int *ii, long long *ii_squared,
		int *featureOffsets, float *ensemblePosteriors, CvRect *fgList,
		unsigned char *imgData, float *posteriors, float *variances,
		int *indexes) {

	int i = threadIdx.x + blockIdx.x * blockDim.x;
	if (i < (*info).numWindows) {
		if ((*info).isActive) {
			bool isInside = false;
			for (int j = 0; j < (*info).fgListSize; ++j) {
				int bgBox[4];
				bgBox[0] = fgList[j].x;
				bgBox[1] = fgList[j].y;
				bgBox[2] = fgList[j].width;
				bgBox[3] = fgList[j].height;

				//TODO: This is inefficient and should be replaced by a quadtree
				if (data[i].window[0] > bgBox[0] &&
					 data[i].window[1] > bgBox[1] &&
					 data[i].window[0] + data[i].window[2] < bgBox[0] + bgBox[2] &&
					 data[i].window[1] + data[i].window[3] < bgBox[1] + bgBox[3])
					isInside = true;
			}

			if (!isInside) {
				posteriors[i] = 0;
				indexes[i] = -1;
				return;
			}
		}

		// executing varianceFilter
		float mX = (ii[data[i].varianceWindowOffsets[3]] -
							ii[data[i].varianceWindowOffsets[2]] -
							ii[data[i].varianceWindowOffsets[1]] +
							ii[data[i].varianceWindowOffsets[0]]) /
							(float) data[i].varianceWindowOffsets[5]; //Sum of Area divided by area
		float mX2 = (ii_squared[data[i].varianceWindowOffsets[3]] -
							  ii_squared[data[i].varianceWindowOffsets[2]] -
							  ii_squared[data[i].varianceWindowOffsets[1]] +
							  ii_squared[data[i].varianceWindowOffsets[0]]) /
							  (float) data[i].varianceWindowOffsets[5];
		float bboxvar = mX2 - mX * mX;
		variances[i] = bboxvar;
		if (bboxvar < 0) {
			posteriors[i] = 0;
			indexes[i] = -1;
			return;
		}

		// executing ensembleClassifier
		float conf = 0.0;
		for(int n = 0; n < TLD_NUM_TREES; ++n) {
			int index = 0;
			int *off = featureOffsets + data[i].ensembleWindowOffsets[4]
					+ n * 2 * TLD_NUM_FEATURES; //bbox[4] is pointer to features for the current scale

			for (int j = 0; j < TLD_NUM_FEATURES; ++j) {
				index <<= 1;

				int fp0 = imgData[data[i].ensembleWindowOffsets[0] + off[0]];
				int fp1 = imgData[data[i].ensembleWindowOffsets[0] + off[1]];

				if (fp0 > fp1) {
					index |= 1;
				}

				off += 2;
			}

			data[i].featureVector[n] = index;
			int indice = pow(2.0f, 13);
			conf += ensemblePosteriors[n * indice + data[i].featureVector[n]];
		}
		posteriors[i] = conf;

		if (posteriors[i] < 0.5) {
			indexes[i] = -1;
			return;
		}

		// executing nnClassifier
		float mean = 0;
		for (int k = 0; k < TLD_PATCH_SIZE; ++k) {
			for (int l = 0; l < TLD_PATCH_SIZE; ++l) {
				mean += data[i].subImgData[l * data[i].step + k];
			}
		}

		mean /= TLD_PATCH_SIZE * TLD_PATCH_SIZE;

		float values[TLD_PATCH_SIZE * TLD_PATCH_SIZE];
		for (int k = 0; k < TLD_PATCH_SIZE; ++k) {
			for (int l = 0; l < TLD_PATCH_SIZE; ++l) {
				values[l * TLD_PATCH_SIZE + k] = data[i].subImgData[l * data[i].step + k] - mean;
			}
		}

		conf = 0;
		int size = TLD_PATCH_SIZE * TLD_PATCH_SIZE;

		float ccorr_max_p = 0;

		//Compare patch to positive patches
		for (int j = 0; j < info->truePositiveSize; ++j) {
			double corr = 0;
			double norm1 = 0;
			double norm2 = 0;

			for (int k = 0; k < size; ++k) {
				corr += truePositives[k] * values[k];
				norm1 += truePositives[k] * truePositives[k];
				norm2 += values[k] * values[k];
			}

			// normalization to <0,1>

			float ccorr = (corr / sqrt(norm1 * norm2) + 1) / 2.0;

			if (ccorr > ccorr_max_p) {
				ccorr_max_p = ccorr;
			}
		}

		float ccorr_max_n = 0;

		//Compare patch to negative patches
		for (int j = 0; j < info->falsePositiveSize; ++j) {
			double corr = 0;
			double norm1 = 0;
			double norm2 = 0;

			for (int k = 0; k < size; ++k) {
				corr += falsePositives[k] * values[k];
				norm1 += falsePositives[k] * falsePositives[k];
				norm2 += values[k] * values[k];
			}

			// normalization to <0,1>

			float ccorr = (corr / sqrt(norm1 * norm2) + 1) / 2.0;

			if (ccorr > ccorr_max_n) {
				ccorr_max_n = ccorr;
			}
		}

		float dN = 1 - ccorr_max_n;
		float dP = 1 - ccorr_max_p;

		conf = dN / (dN + dP);

		if (conf < 0.65) {
			indexes[i] = -1;
			return;
		}
	}
}

void safeCall(hipError_t e) {
	if (e != hipSuccess) {
		printf("CUDA error: %s\n", hipGetErrorString(e));
		system("pause");
	}
}

void detectionWithCuda(DetectorCascade *detectorCascade, const Mat &img, detectionData &detectCuda) {

	int numWindows = detectorCascade->numWindows;
	int numScales = detectorCascade->numScales;
	int positiveSampleSize = TLD_PATCH_SIZE * TLD_PATCH_SIZE;
	int falsePositivesSize = detectCuda.falsePositives->size();
	int truePositivesSize = detectCuda.truePositives->size();
	int numImgBytes = img.total() * img.elemSize();

	// assignment
	refInfo info;
	info.numWindows = detectorCascade->numWindows;
	info.isActive = detectorCascade->foregroundDetector->isActive();
	info.fgListSize = detectCuda.fgList->size();
	info.falsePositiveSize = falsePositivesSize;
	info.truePositiveSize = truePositivesSize;

	int *index = new int[numWindows];
	proData *data = new proData[numWindows];
	for (int i = 0; i < numWindows; ++i) {
		int *window = &detectorCascade->windows[TLD_WINDOW_SIZE * i];
		for (int j = 0; j < TLD_WINDOW_SIZE; ++j)
			data[i].window[j] = window[j];

		int *windowOffsets = &detectorCascade->windowOffsets[TLD_WINDOW_OFFSET_SIZE * i];
		for (int k = 0; k < TLD_WINDOW_OFFSET_SIZE; ++k) {
			data[i].varianceWindowOffsets[k] = windowOffsets[k];
			data[i].ensembleWindowOffsets[k] = windowOffsets[k];
		}
		for (int l = 0; l < TLD_NUM_TREES; ++l)
			data[i].featureVector[l] = 0;

		int x = data[i].window[0];
		int y = data[i].window[1];
		int w = data[i].window[2];
		int h = data[i].window[3];
		Mat subImage;
		subImage = img(Rect(x, y, w, h)).clone();
		Mat result;
		resize(subImage, result, cvSize(TLD_PATCH_SIZE, TLD_PATCH_SIZE)); //Default is bilinear
		for (int n = 0; n < positiveSampleSize; ++n)
			data[i].subImgData[n] = result.data[n];
		data[i].step = result.step;

		index[i] = i;
	}
	float *falsePositives = new float[falsePositivesSize * positiveSampleSize];
	float *truePositives = new float[truePositivesSize * positiveSampleSize];
	for (int i = 0; i < falsePositivesSize; ++i)
		for (int j = 0; j < positiveSampleSize; ++j)
			falsePositives[i * falsePositivesSize + j] = detectCuda.falsePositives->at(i).values[j];
	for (int i = 0; i < truePositivesSize; ++i)
		for (int j = 0; j < positiveSampleSize; ++j)
			truePositives[i * truePositivesSize + j] = detectCuda.truePositives->at(i).values[j];

	CvRect *fgList = new CvRect[detectCuda.fgList->size()];
	for (size_t i = 0; i < detectCuda.fgList->size(); ++i) {
		fgList[i].x = detectCuda.fgList->at(i).x;
		fgList[i].y = detectCuda.fgList->at(i).y;
		fgList[i].width = detectCuda.fgList->at(i).width;
		fgList[i].height = detectCuda.fgList->at(i).height;
	}

	// declare device variables
	refInfo *dev_info = NULL;
	proData *dev_data = NULL;
	float *dev_falsePositives = NULL;
	float *dev_truePositives = NULL;
	int *dev_ii = NULL;
	long long *dev_ii_squared = NULL;
	int *dev_featureOffsets = NULL;
	float *dev_ensemblePosteriors = NULL;
	CvRect *dev_fgList = NULL;
	unsigned char *dev_imgData = NULL;
	float *dev_posteriors = NULL;
	float *dev_variances = NULL;
	int *dev_indexes = NULL;

	// allocate GPU memory for input data
	safeCall(hipMalloc((void**) &dev_info, sizeof(refInfo)));
	safeCall(hipMalloc((void**) &dev_data, sizeof(proData)));
	safeCall(hipMalloc((void**) &dev_falsePositives, falsePositivesSize * positiveSampleSize * sizeof(float)));
	safeCall(hipMalloc((void**) &dev_truePositives, truePositivesSize * positiveSampleSize * sizeof(float)));
	safeCall(hipMalloc((void**) &dev_ii, img.size().width * img.size().height * sizeof(int)));
	safeCall(hipMalloc((void**) &dev_ii_squared, img.size().width * img.size().height * sizeof(long long)));
	safeCall(hipMalloc((void**) &dev_featureOffsets, numScales * TLD_NUM_TREES * TLD_NUM_FEATURES * 2 * sizeof(int)));
	safeCall(hipMalloc((void**) &dev_ensemblePosteriors, TLD_NUM_TREES * TLD_NUM_INDICES * sizeof(float)));
	safeCall(hipMalloc((void**) &dev_fgList, detectCuda.fgList->size() * sizeof(CvRect)));
	safeCall(hipMalloc((void**) &dev_imgData, numImgBytes));

	// copy input data to GPU
	safeCall(hipMemcpy(dev_info, &info, sizeof(refInfo), hipMemcpyHostToDevice));
	safeCall(hipMemcpy(dev_data, data, numWindows * sizeof(proData), hipMemcpyHostToDevice));
	safeCall(hipMemcpy(dev_falsePositives, falsePositives, falsePositivesSize * positiveSampleSize * sizeof(float), hipMemcpyHostToDevice));
	safeCall(hipMemcpy(dev_truePositives, truePositives, truePositivesSize * positiveSampleSize * sizeof(float), hipMemcpyHostToDevice));
	safeCall(hipMemcpy(dev_ii, detectCuda.ii, img.size().width * img.size().height * sizeof(int), hipMemcpyHostToDevice));
	safeCall(hipMemcpy(dev_ii_squared, detectCuda.ii_squared, img.size().width * img.size().height * sizeof(long long), hipMemcpyHostToDevice));
	safeCall(hipMemcpy(dev_featureOffsets, detectCuda.featureOffsets, numScales * TLD_NUM_TREES * TLD_NUM_FEATURES * 2 * sizeof(int), hipMemcpyHostToDevice));
	safeCall(hipMemcpy(dev_ensemblePosteriors, detectCuda.ensemblePosteriors, TLD_NUM_TREES * TLD_NUM_INDICES * sizeof(float), hipMemcpyHostToDevice));
	safeCall(hipMemcpy(dev_fgList, fgList, detectCuda.fgList->size() * sizeof(CvRect), hipMemcpyHostToDevice));
	safeCall(hipMemcpy(dev_imgData, (const unsigned char *)img.data, numImgBytes, hipMemcpyHostToDevice));

	// allocate GPU memory for output data
	safeCall(hipMalloc((void**) &dev_posteriors, numWindows * sizeof(float)));
	safeCall(hipMalloc((void**) &dev_variances, numWindows * sizeof(float)));
	safeCall(hipMalloc((void**) &dev_indexes, numWindows * sizeof(int)));
	safeCall(hipMemcpy(dev_indexes, index, numWindows * sizeof(int), hipMemcpyHostToDevice));

	// invoke cuda parallel computing
	detection<<<1024, 1024>>>(dev_info, dev_data, dev_falsePositives, dev_truePositives,
			dev_ii, dev_ii_squared, dev_featureOffsets, dev_ensemblePosteriors,  dev_fgList, dev_imgData, dev_posteriors,
			dev_variances, dev_indexes);

	float *posteriors = new float[numWindows];
	safeCall(hipMemcpy(posteriors, dev_posteriors, numWindows * sizeof(float), hipMemcpyDeviceToHost));
	float *variances = new float[numWindows];
	safeCall(hipMemcpy(variances, dev_variances, numWindows * sizeof(float), hipMemcpyDeviceToHost));

	safeCall(hipMemcpy(index, dev_indexes, numWindows * sizeof(int), hipMemcpyDeviceToHost));

	for (int i = 0; i < numWindows; ++i) {
		detectCuda.posteriors[i] = posteriors[i];
		detectCuda.variances[i] = variances[i];
		if (index[i] >= 0)
			detectCuda.confidentIndices->push_back(index[i]);
	}

	safeCall(hipFree(dev_info));
	safeCall(hipFree(dev_data));
	safeCall(hipFree(dev_falsePositives));
	safeCall(hipFree(dev_truePositives));
	safeCall(hipFree(dev_ii));
	safeCall(hipFree(dev_ii_squared));
	safeCall(hipFree(dev_featureOffsets));
	safeCall(hipFree(dev_ensemblePosteriors));
	safeCall(hipFree(dev_fgList));
	safeCall(hipFree(dev_imgData));
	safeCall(hipFree(dev_posteriors));
	safeCall(hipFree(dev_variances));
	safeCall(hipFree(dev_indexes));

	delete posteriors;
	delete variances;
	delete fgList;
	delete truePositives;
	delete falsePositives;
	delete data;
	delete index;

	return;
}

/*struct result{
	int fern[10];
	int res;
	float conf;
};

struct proData{
	int size, nstructs, structSize, featureSizeSize;
	float var, thr_fern;
};

struct boundingbox{
	int x;
	int y;
	int width; 
	int height;
	int sidx;
};
struct feature{
	uchar x1, y1, x2, y2;
};

__global__ void kernel(float* posteriors, feature* featureData, proData * dt, boundingbox* bbdata, const PtrStepSz<uchar1> img, const PtrStepSz<float1> iisum, PtrStep<float1> iisqsum, result *result)
{
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	if (i < (*dt).size){
		result[i].res = 0;
		int fern[10];
		feature featureTmp;
		float brs = iisum(bbdata[i].y + bbdata[i].height, bbdata[i].x + bbdata[i].width).x;
		float bls = iisum(bbdata[i].y + bbdata[i].height, bbdata[i].x).x;
		float trs = iisum(bbdata[i].y, bbdata[i].x + bbdata[i].width).x;
		float tls = iisum(bbdata[i].y, bbdata[i].x).x;
		float brsq = iisqsum(bbdata[i].y + bbdata[i].height, bbdata[i].x + bbdata[i].width).x;
		float blsq = iisqsum(bbdata[i].y + bbdata[i].height, bbdata[i].x).x;
		float trsq = iisqsum(bbdata[i].y, bbdata[i].x + bbdata[i].width).x;
		float tlsq = iisqsum(bbdata[i].y, bbdata[i].x).x;
		float mean = (brs + tls - trs - bls) / ((float)bbdata[i].height*(float)bbdata[i].width);
		float sqmean = (brsq + tlsq - trsq - blsq) / ((float)bbdata[i].height*(float)bbdata[i].width);
		if (sqmean - mean*mean >= (*dt).var){
			int leaf;
			for (int t = 0; t<(*dt).nstructs; t++){
				leaf = 0;
				for (int f = 0; f<(*dt).structSize; f++){
					featureTmp = featureData[bbdata[i].sidx*(*dt).featureSizeSize + t*(*dt).structSize + f];
					leaf = (leaf << 1) + (img(bbdata[i].y + featureTmp.y1, bbdata[i].x + featureTmp.x1).x > img(bbdata[i].y+featureTmp.y2, bbdata[i].x+featureTmp.x2).x);
				}
				fern[t] = leaf;
			}
			float votes = 0;
			for (int t = 0; t <(*dt).nstructs; t++) {
				votes += posteriors[t*(int)pow(2.0f, (float)(*dt).structSize)+ fern[t]];
			}
			result[i].conf = votes;
			for (int t = 0; t<10; t++){
				result[i].fern[t] = fern[t];
			}
			if (votes>(*dt).nstructs*(*dt).thr_fern){
				result[i].res = 1;
			}
		}
		else{
			result[i].conf = 0.0;
		}
	}
}

void safeCall(hipError_t e){
	if (e != hipSuccess){
		printf("CUDA error: %s\n", hipGetErrorString(e));
		system("pause");
	}
}

void processWithCuda(TLD *tld, const Mat &img, const FerNNClassifier &classifier){
	GpuMat gpuiisum, gpuiisqsum, gpuimg;
	gpuiisum.upload(tld->iisum);
	gpuiisum.convertTo(gpuiisum, CV_32F);
	gpuiisqsum.upload(tld->iisqsum);
	gpuiisqsum.convertTo(gpuiisqsum, CV_32F);
	gpuimg.upload(img);
	vector<BoundingBox> grid = tld->grid;
	proData dt;
	dt.size = grid.size();
	dt.var = tld->var;
	dt.nstructs = classifier.nstructs;
	dt.structSize = classifier.structSize;
	dt.thr_fern = classifier.thr_fern;
	boundingbox * data = new boundingbox[dt.size];
	for (int i = 0; i < dt.size; i++){
		data[i].x = grid[i].x;
		data[i].y = grid[i].y;
		data[i].width = grid[i].width;
		data[i].height = grid[i].height;
		data[i].sidx = grid[i].sidx;
	}
	int featureSize = classifier.features.size();
	int featureSizeSize = classifier.features[0].size();
	dt.featureSizeSize = featureSizeSize;
	feature * featureData = new feature[featureSize*featureSizeSize];
	for (int i = 0; i < featureSize; i++){
		for (int j = 0; j < featureSizeSize; j++){
			featureData[i*featureSizeSize + j].x1 = classifier.features[i][j].x1;
			featureData[i*featureSizeSize + j].y1 = classifier.features[i][j].y1;
			featureData[i*featureSizeSize + j].x2 = classifier.features[i][j].x2;
			featureData[i*featureSizeSize + j].y2 = classifier.features[i][j].y2;
		}
	}
	boundingbox *dev_bbdata = NULL;
	feature *dev_featureData = NULL;
	proData *dev_dt = NULL;
	float * dev_posteriors = NULL;
	int posteriorsSize = classifier.posteriors.size();
	int posteriorsSizeSize = classifier.posteriors[0].size();
	float *posteriors = new float[posteriorsSize*posteriorsSizeSize];
	for (int i = 0; i < posteriorsSize; i++){
		for (int j = 0; j < posteriorsSizeSize; j++){
			posteriors[i*posteriorsSizeSize + j] = classifier.posteriors[i][j];
		}
	}
	safeCall(hipMalloc((void**)&dev_posteriors, posteriorsSize*posteriorsSizeSize*sizeof(float)));
	safeCall(hipMalloc((void**)&dev_featureData, featureSize*featureSizeSize*sizeof(feature)));
	safeCall(hipMalloc((void**)&dev_bbdata, dt.size*sizeof(boundingbox)));
	safeCall(hipMalloc((void**)&dev_dt, sizeof(proData)));
	safeCall(hipMemcpy(dev_posteriors, posteriors, posteriorsSize*posteriorsSizeSize*sizeof(float), hipMemcpyHostToDevice));
	safeCall(hipMemcpy(dev_dt, &dt, sizeof(proData), hipMemcpyHostToDevice));
	safeCall(hipMemcpy(dev_bbdata, data, dt.size*sizeof(boundingbox), hipMemcpyHostToDevice));
	safeCall(hipMemcpy(dev_featureData, featureData, featureSize*featureSizeSize*sizeof(feature), hipMemcpyHostToDevice));

	result *dev_result = NULL;
	result *res = new result[dt.size];
	safeCall(hipMalloc((void**)&dev_result, dt.size*sizeof(result)));
	kernel << <1024, 1024 >> >(dev_posteriors, dev_featureData, dev_dt, dev_bbdata, gpuimg, gpuiisum, gpuiisqsum, dev_result);
	safeCall(hipMemcpy(res, dev_result, dt.size * sizeof(result), hipMemcpyDeviceToHost));

	int output = 0;
	for (int i = 0; i < dt.size; i++){
		if (res[i].res)
			output++;
	}
	printf("%d %d\n", dt.size, output);

	for (int i = 0; i < dt.size; i++){
		tld->tmp.conf[i] = res[i].conf;
		for (int j = 0; j < 10; j++){
			tld->tmp.patt[i][j] = res[i].fern[j];
		}
		if (res[i].res){
			tld->dt.bb.push_back(i);
		}
	}

	safeCall(hipFree(dev_posteriors));
	safeCall(hipFree(dev_bbdata));
	safeCall(hipFree(dev_dt));
	safeCall(hipFree(dev_featureData));
	safeCall(hipFree(dev_result));
	
	delete data;
	delete featureData;
	delete res;

	return;
}*/
